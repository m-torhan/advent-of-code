#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <iostream>
#include <vector>

constexpr int N = 32;
constexpr dim3 blockDim(16, 16);
constexpr dim3 gridDim(N / blockDim.x, N / blockDim.y);
constexpr int blocksPerGrid = gridDim.x * gridDim.y;

__global__ void find_guard(char *map, int *mapShape, int *guardPos) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int pos[2];
    bool found = false;

    while (x < mapShape[1]) {
        while (y < mapShape[0]) {
            if (('^' == map[x + mapShape[1] * y]) || ('>' == map[x + mapShape[1] * y]) ||
                ('v' == map[x + mapShape[1] * y]) || ('<' == map[x + mapShape[1] * y])) {
                pos[0] = x;
                pos[1] = y;
                found = true;
            }
            y += blockDim.y * gridDim.y;
        }
        x += blockDim.x * gridDim.x;
        y = threadIdx.y + blockIdx.y * blockDim.y;
    }

    __syncthreads();

    if (found) {
        guardPos[0] = pos[0];
        guardPos[1] = pos[1];
    }
}

__global__ void count_tiles(char *map, int *mapShape, int *count) {
    __shared__ int cache[blockDim.x * blockDim.y];

    int cacheIdx = threadIdx.x + threadIdx.y * blockDim.x;
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int partialCount = 0;

    while (x < mapShape[1]) {
        while (y < mapShape[0]) {
            partialCount += ('x' == map[x + mapShape[1] * y]);
            y += blockDim.y * gridDim.y;
        }
        x += blockDim.x * gridDim.x;
        y = threadIdx.y + blockIdx.y * blockDim.y;
    }
    cache[cacheIdx] = partialCount;

    __syncthreads();

    int idx = (blockDim.x * blockDim.y) / 2;
    while (idx != 0) {
        if (cacheIdx < idx) {
            cache[cacheIdx] += cache[cacheIdx + idx];
        }
        __syncthreads();
        idx /= 2;
    }

    if (cacheIdx == 0) {
        count[blockIdx.x + blockIdx.y * gridDim.x] = cache[0];
    }
}

enum class Dir {
    UP,
    RIGHT,
    LEFT,
    DOWN,
};

void move_guard(char *map, const int mapShape[2], const int guardPos[2]) {
    int x = guardPos[0];
    int y = guardPos[1];

    Dir d;

    switch (map[x + y * mapShape[1]]) {
    case '^':
        d = Dir::UP;
        break;
    case '>':
        d = Dir::RIGHT;
        break;
    case 'v':
        d = Dir::DOWN;
        break;
    case '<':
        d = Dir::LEFT;
        break;
    }

    while (0 <= x && x <= mapShape[0] && 0 <= y && y <= mapShape[1]) {
        int x_next = x;
        int y_next = y;
        switch (d) {
        case Dir::UP:
            --y_next;
            break;
        case Dir::RIGHT:
            ++x_next;
            break;
        case Dir::DOWN:
            ++y_next;
            break;
        case Dir::LEFT:
            --x_next;
            break;
        }
        if (x_next < 0 || x_next >= mapShape[0] || y_next < 0 || y_next >= mapShape[1]) {
            map[x + y * mapShape[1]] = 'x';
            break;
        }
        if (map[x_next + y_next * mapShape[1]] != '#') {
            map[x + y * mapShape[1]] = 'x';
            x = x_next;
            y = y_next;
        } else {
            switch (d) {
            case Dir::UP:
                d = Dir::RIGHT;
                break;
            case Dir::RIGHT:
                d = Dir::DOWN;
                break;
            case Dir::DOWN:
                d = Dir::LEFT;
                break;
            case Dir::LEFT:
                d = Dir::UP;
                break;
            }
        }
    }
}

int main() {
    std::vector<std::vector<char>> mapVec;

    std::string line;

    while (std::getline(std::cin, line)) {
        std::vector<char> lineVec(line.begin(), line.end());
        mapVec.push_back(lineVec);
    }

    int mapShape[2] = {(int)mapVec.size(), (int)mapVec[0].size()};
    int mapSize = mapShape[0] * mapShape[1];
    char *map = new char[mapSize];
    int guardPos[2];
    int *partialCount = new int[blocksPerGrid];

    for (int i = 0; i < mapShape[1]; ++i) {
        memcpy(&map[i * mapShape[0]], mapVec[i].data(), mapShape[1] * sizeof(char));
    }

    char *dev_map;
    int *dev_mapShape;
    int *dev_partialCount;
    int *dev_guardPos;

    CUDA_CHECK(hipMalloc((void **)&dev_map, mapSize * sizeof(char)));
    CUDA_CHECK(hipMalloc((void **)&dev_mapShape, 2 * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_guardPos, 2 * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_partialCount, blocksPerGrid * sizeof(int)));

    CUDA_CHECK(hipMemcpy(dev_map, map, mapSize * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_mapShape, mapShape, 2 * sizeof(int), hipMemcpyHostToDevice));

    find_guard<<<gridDim, blockDim>>>(dev_map, dev_mapShape, dev_guardPos);

    CUDA_CHECK(hipMemcpy(guardPos, dev_guardPos, 2 * sizeof(int), hipMemcpyDeviceToHost));

    move_guard(map, mapShape, guardPos);

    CUDA_CHECK(hipMemcpy(dev_map, map, mapSize * sizeof(char), hipMemcpyHostToDevice));

    count_tiles<<<gridDim, blockDim>>>(dev_map, dev_mapShape, dev_partialCount);

    CUDA_CHECK(hipMemcpy(partialCount, dev_partialCount, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));

    int count = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        count += partialCount[i];
    }

    std::cout << "Tiles count: " << count << std::endl;

    hipFree(dev_map);
    hipFree(dev_mapShape);
    hipFree(dev_partialCount);
    hipFree(dev_guardPos);

    delete[] map;
    delete[] partialCount;
}
