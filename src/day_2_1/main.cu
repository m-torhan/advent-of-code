#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <iostream>
#include <sstream>
#include <vector>

constexpr int N = 1024;
constexpr int threadsPerBlock = 256;
constexpr int blocksPerGrid = std::min(32, 1 + (N - 1) / threadsPerBlock);

__device__ bool is_report_safe(int *report, int reportLen) {
    bool ret = true;

    for (int i = 0; i < reportLen - 2; ++i) {
        if ((report[i] < report[i + 1] && report[i + 1] > report[i + 2]) ||
            (report[i] > report[i + 1] && report[i + 1] < report[i + 2])) {
            ret = false;
        }
    }
    for (int i = 0; i < reportLen - 1; ++i) {
        int delta = abs(report[i + 1] - report[i]);
        if (delta < 1 || delta > 3) {
            ret = false;
        }
    }

    return ret;
}

__global__ void are_reports_safe(int *report, int *reportLen, int *safe, int maxReportLen, size_t size) {
    __shared__ int cache[threadsPerBlock];

    int reportIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIdx = threadIdx.x;

    int partial = 0;
    while (reportIdx < size) {
        partial += is_report_safe(&report[reportIdx * maxReportLen], reportLen[reportIdx]);
        reportIdx += blockDim.x * gridDim.x;
    }

    cache[cacheIdx] = partial;

    __syncthreads();

    int idx = blockDim.x / 2;
    while (idx != 0) {
        if (cacheIdx < idx) {
            cache[cacheIdx] += cache[cacheIdx + idx];
        }
        __syncthreads();
        idx /= 2;
    }

    if (cacheIdx == 0) {
        safe[blockIdx.x] = cache[0];
    }
}

int main() {
    std::vector<std::vector<int>> reportsVec;

    size_t maxReportLen = 0;

    for (std::string line; std::getline(std::cin, line);) {
        std::stringstream sline(line);
        reportsVec.push_back({});

        int num;
        while (sline >> num) {
            reportsVec.back().push_back(num);
        }
        maxReportLen = std::max(maxReportLen, reportsVec.back().size());
    }

    const auto reportsNum = reportsVec.size();

    int *reports = new int[maxReportLen * reportsNum];
    int *reportsLen = new int[reportsNum];

    for (int i = 0; i < reportsNum; ++i) {
        reportsLen[i] = reportsVec[i].size();
        memcpy(&reports[i * maxReportLen], reportsVec[i].data(), reportsVec[i].size() * sizeof(int));
    }

    auto *partialSum = new int[blocksPerGrid];
    int *dev_reports, *dev_reportsLen, *dev_partialSum;

    CUDA_CHECK(hipMalloc((void **)&dev_reports, maxReportLen * reportsNum * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_reportsLen, reportsNum * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_partialSum, blocksPerGrid * sizeof(int)));

    CUDA_CHECK(hipMemcpy(dev_reports, reports, maxReportLen * reportsNum * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_reportsLen, reportsLen, reportsNum * sizeof(int), hipMemcpyHostToDevice));

    are_reports_safe<<<blocksPerGrid, threadsPerBlock>>>(dev_reports, dev_reportsLen, dev_partialSum, maxReportLen,
                                                         reportsNum);

    CUDA_CHECK(hipMemcpy(partialSum, dev_partialSum, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));

    int sum = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        sum += partialSum[i];
    }

    std::cout << "Safe reports count: " << sum << std::endl;

    hipFree(dev_reports);
    hipFree(dev_reportsLen);
    hipFree(dev_partialSum);

    delete[] reports;
    delete[] reportsLen;
}
