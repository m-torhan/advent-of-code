#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <iostream>
#include <vector>

constexpr int N = 32;
constexpr dim3 blockDim(16, 16);
constexpr dim3 gridDim(N / blockDim.x, N / blockDim.y);
constexpr int blocksPerGrid = gridDim.x * gridDim.y;

__device__ int gcd(int a, int b) {
    int r = 0;
    a = abs(a);
    b = abs(b);
    while (b) {
        r = a % b;
        a = b;
        b = r;
    }
    return a;
}

__global__ void find_antinodes(const char *antennas, const int *antennasPos, int antennasCount, int mapShapeX,
                               int mapShapeY, int *antinodes) {
    for (int x = threadIdx.x + blockIdx.x * blockDim.x; x < antennasCount; x += blockDim.x * gridDim.x) {
        for (int y = threadIdx.y + blockIdx.y * blockDim.y; y < antennasCount; y += blockDim.y * gridDim.y) {
            if (x < y && antennas[x] == antennas[y]) {
                int deltaX = antennasPos[2 * y] - antennasPos[2 * x];
                int deltaY = antennasPos[2 * y + 1] - antennasPos[2 * x + 1];

                int deltaGCD = gcd(deltaX, deltaY);

                deltaX /= deltaGCD;
                deltaY /= deltaGCD;

                for (int i = 0;; ++i) {
                    int antinodeX = antennasPos[2 * x] + i * deltaX;
                    int antinodeY = antennasPos[2 * x + 1] + i * deltaY;
                    if (0 <= antinodeX && antinodeX < mapShapeX && 0 <= antinodeY && antinodeY < mapShapeY) {
                        antinodes[antinodeX + antinodeY * mapShapeX] = 1;
                    } else {
                        break;
                    }
                }

                for (int i = 1;; ++i) {
                    int antinodeX = antennasPos[2 * x] - i * deltaX;
                    int antinodeY = antennasPos[2 * x + 1] - i * deltaY;
                    if (0 <= antinodeX && antinodeX < mapShapeX && 0 <= antinodeY && antinodeY < mapShapeY) {
                        antinodes[antinodeX + antinodeY * mapShapeX] = 1;
                    } else {
                        break;
                    }
                }
            }
        }
    }
}
__global__ void count_antinodes(int *antinodes, int mapShapeX, int mapShapeY, int *count) {
    __shared__ int cache[blockDim.x * blockDim.y];

    int cacheIdx = threadIdx.x + threadIdx.y * blockDim.x;

    int partialCount = 0;

    for (int x = threadIdx.x + blockIdx.x * blockDim.x; x < mapShapeX; x += blockDim.x * gridDim.x) {
        for (int y = threadIdx.y + blockIdx.y * blockDim.y; y < mapShapeY; y += blockDim.y * gridDim.y) {
            partialCount += antinodes[x + y * mapShapeX];
        }
    }
    cache[cacheIdx] = partialCount;

    __syncthreads();

    int idx = (blockDim.x * blockDim.y) / 2;
    while (idx != 0) {
        if (cacheIdx < idx) {
            cache[cacheIdx] += cache[cacheIdx + idx];
        }
        __syncthreads();
        idx /= 2;
    }

    if (cacheIdx == 0) {
        count[blockIdx.x + blockIdx.y * gridDim.x] = cache[0];
    }
}

int main() {
    std::vector<std::vector<char>> mapVec;

    std::string line;

    while (std::getline(std::cin, line)) {
        std::vector<char> lineVec(line.begin(), line.end());
        mapVec.push_back(lineVec);
    }

    int mapShape[2] = {(int)mapVec[0].size(), (int)mapVec.size()};
    int mapSize = mapShape[0] * mapShape[1];

    std::vector<char> antennasVec;
    std::vector<std::pair<int, int>> antennasPosVec;

    for (int x = 0; x < mapShape[0]; ++x) {
        for (int y = 0; y < mapShape[1]; ++y) {
            char freq = mapVec[y][x];
            if (freq != '.') {
                antennasVec.push_back(freq);
                antennasPosVec.push_back({x, y});
            }
        }
    }

    const int antennasCount = antennasVec.size();
    int *antennasPos = new int[antennasCount * 2];
    int *partialCount = new int[blocksPerGrid];

    for (int i = 0; i < antennasCount; ++i) {
        antennasPos[2 * i] = antennasPosVec[i].first;
        antennasPos[2 * i + 1] = antennasPosVec[i].second;
    }

    char *dev_antennas;
    int *dev_antennasPos;
    int *dev_antinodes;
    int *dev_partialCount;

    CUDA_CHECK(hipMalloc((void **)&dev_antennas, antennasCount * sizeof(char)));
    CUDA_CHECK(hipMalloc((void **)&dev_antennasPos, antennasCount * 2 * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_antinodes, mapSize * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_partialCount, blocksPerGrid * sizeof(int)));

    CUDA_CHECK(hipMemcpy(dev_antennas, antennasVec.data(), antennasCount * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_antennasPos, antennasPos, 2 * antennasCount * sizeof(int), hipMemcpyHostToDevice));

    find_antinodes<<<gridDim, blockDim>>>(dev_antennas, dev_antennasPos, antennasCount, mapShape[0], mapShape[1],
                                          dev_antinodes);

    count_antinodes<<<gridDim, blockDim>>>(dev_antinodes, mapShape[0], mapShape[1], dev_partialCount);

    CUDA_CHECK(hipMemcpy(partialCount, dev_partialCount, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));

    int count = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        count += partialCount[i];
    }

    std::cout << "Antinodes count: " << count << std::endl;

    hipFree(dev_antennas);
    hipFree(dev_antennasPos);
    hipFree(dev_antinodes);
    hipFree(dev_partialCount);

    delete[] antennasPos;
    delete[] partialCount;
}
