#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <iostream>
#include <vector>

constexpr int N = 1024;
constexpr dim3 blockDim(16, 16);
constexpr dim3 gridDim(N / blockDim.x, N / blockDim.y);
constexpr int blocksPerGrid = gridDim.x * gridDim.y;

__global__ void similarity_score(int *a, int *b, int *c, size_t size) {
    __shared__ int cache[blockDim.x * blockDim.y];

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int cacheIdx = threadIdx.x + threadIdx.y * blockDim.x;

    int partial = 0;
    while (x < size) {
        while (y < size) {
            if (a[x] == b[y]) {
                partial += a[x];
            }
            y += blockDim.y * gridDim.y;
        }
        x += blockDim.x * gridDim.x;
    }

    cache[cacheIdx] = partial;

    __syncthreads();

    int idx = (blockDim.x * blockDim.y) / 2;
    while (idx != 0) {
        if (cacheIdx < idx) {
            cache[cacheIdx] += cache[cacheIdx + idx];
        }
        __syncthreads();
        idx /= 2;
    }

    if (cacheIdx == 0) {
        c[blockIdx.x + blockIdx.y * gridDim.x] = cache[0];
    }
}

int main() {
    std::vector<int> left;
    std::vector<int> right;

    int leftNum;
    int rightNum;

    while (std::cin >> leftNum >> rightNum) {
        left.push_back(leftNum);
        right.push_back(rightNum);
    }

    const auto vecLength = left.size();

    auto *partialScore = new int[blocksPerGrid];
    int *dev_left, *dev_right, *dev_partialScore;

    CUDA_CHECK(hipMalloc((void **)&dev_left, vecLength * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_right, vecLength * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&dev_partialScore, blocksPerGrid * sizeof(int)));

    CUDA_CHECK(hipMemcpy(dev_left, left.data(), vecLength * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_right, right.data(), vecLength * sizeof(int), hipMemcpyHostToDevice));

    similarity_score<<<gridDim, blockDim>>>(dev_left, dev_right, dev_partialScore, vecLength);

    CUDA_CHECK(hipMemcpy(partialScore, dev_partialScore, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));

    int score = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        score += partialScore[i];
    }

    std::cout << "Similarity score: " << score << std::endl;

    hipFree(dev_left);
    hipFree(dev_right);
    hipFree(dev_partialScore);

    delete[] partialScore;
}
